#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<>
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include<math.h>

__global__ void sobel(unsigned char *inputImage, unsigned char *output, int width, int height,float *gradientMagnitude) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int sobel_x[5][5] = { {-1,-2,0,2,1},{-4,-8,0,8,4},{-16,-12,0,12,16},{-4,-8,0,8,4},{-1,-2,0,2,1}};
        int sobel_y[5][5] = { {-1,-4,-6,-4,-1},{-2,-8,-12,-8,-2},{0,0,0,0,0},{2,8,12,8,2},{1,4,6,4,1}};
        int sobel_d[5][5] = { {-6,-4,-1,-2,0},{-4,-12,-8,0,2},{-1,-8,0,8,1},{-2,0,8,12,4},{0,2,1,4,6}};
        int sobel_dt[5][5] = { {0,-2,-1,-4,-6},{2,0,-8,-12,-4},{1,8,0,-8,-1},{4,12,8,0,-2},{6,4,1,2,0}};

        float gradient_x = 0.0;
        float gradient_y = 0.0;
        float gradient_d = 0.0;
        float gradient_dt = 0.0;
        for (int i = -2; i <= 2; i++) {
            for (int j = -2; j <= 2; j++) {
              if(x+i>=0 && x+i<width && y+j>=0 && y+j<height){
                gradient_x += sobel_x[i + 2][j + 2] * inputImage[(y + i) * width + (x + j)];
                gradient_y += sobel_y[i + 2][j + 2] * inputImage[(y + i) * width + (x + j)];
                gradient_d += sobel_d[i + 2][j + 2] * inputImage[(y + i) * width + (x + j)];
                gradient_dt += sobel_dt[i + 2][j + 2] * inputImage[(y + i) * width + (x + j)];
              }
            }
        }

        gradientMagnitude[y * width + x] = sqrtf((gradient_x * gradient_x) + (gradient_y * gradient_y) + (gradient_d * gradient_d) + (gradient_dt * gradient_dt));
        if(gradientMagnitude[y * width + x] > 2500){
          output[y*width+x]=255;
        }
        else{
          output[y*width+x]=0;
        }
    }
}

int main() {
    cv::Mat image = cv::imread("input_image.jpg", cv::IMREAD_GRAYSCALE); //change input file name
    if (image.empty()) {
        printf("Error: Unable to load image.\n");
        return -1;
    }
    int width = image.cols;
    int height = image.rows;

    unsigned char *inputImage, *outputImage;
    inputImage = image.data;
    outputImage = (unsigned char*)malloc(width * height * sizeof(unsigned char));

    unsigned char *input, *output;
    hipMalloc((void**)&input, width*height*sizeof(unsigned char));
    hipMalloc((void**)&output, width*height*sizeof(unsigned char));

    float *magnitude;
    hipMalloc((void**)&magnitude, width*height*sizeof(float));

    hipMemcpy(input, inputImage, width*height*sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 blockDim(32, 32);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    sobel<<<gridDim, blockDim>>>(input, output, width, height, magnitude);

    hipError_t cudaErrSync = hipGetLastError();
    if (cudaErrSync != hipSuccess) {
        printf("CUDA Error (Sync): %s\n", hipGetErrorString(cudaErrSync));
        return -1;
    }

    hipMemcpy(outputImage, output, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost);

    cv::Mat processedImage(height, width, CV_8UC1, outputImage);
    cv::imwrite("output5.jpg", processedImage); //change output file name

    hipFree(input);
    hipFree(output);
    free(outputImage);

    return 0;
}
